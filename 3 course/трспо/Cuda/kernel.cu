#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <time.h>
#include <math.h>
#include <chrono>
#include <iomanip>

__global__ void find_foo_kernel(float step, unsigned int n) {
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int stride = gridDim.x * blockDim.x;
	unsigned int offset = 0;
	float x = 0, foo = 0;

	while (index + offset < n) {
		x = 1 + float(index + offset) * (step);
		foo = 2 * (1 / tan(3 * x)) - (1 / (pow(12 * x, 2) + 7 * x - 5));
		offset += stride;
	}

}
__global__ void find_minimum_kernel(const float* array, float* min, int* mutex, unsigned int n) {
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int stride = gridDim.x * blockDim.x;
	unsigned int offset = 0;
	__shared__ float cache[256];

	float temp = RAND_MAX + 1;
	while (index + offset < n) {
		temp = fminf(temp, array[index + offset]);
		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();

	unsigned int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] = fminf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		while (atomicCAS(mutex, 0, 1) == 1) {
			if (*min > cache[0]) {
				*min = cache[0];
			}
		}
		atomicExch(mutex, 0);
	}
}

int main() {
	unsigned int N = pow(10,7);
	float* h_array;
	float* d_array;
	float* h_min;
	float* d_min;
	int* d_mutex;
	h_array = (float*)malloc(N * sizeof(float));
	h_min = (float*)malloc(sizeof(float));
	hipMalloc((void**)&d_array, N * sizeof(float));
	hipMalloc((void**)&d_min, sizeof(float));
	hipMalloc((void**)&d_mutex, sizeof(int));
	hipMemset(d_min, RAND_MAX + 1, sizeof(float));
	hipMemset(d_mutex, 0, sizeof(float));

	srand(time(NULL));
	for (unsigned int i = 0; i < N; i++) {
		h_array[i] = ((double(rand()) / double(RAND_MAX)) * (RAND_MAX - -RAND_MAX)) + -RAND_MAX;
	}

	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipMemcpy(d_array, h_array, N * sizeof(float), hipMemcpyHostToDevice);

	dim3 gridSize = 256;
	dim3 blockSize = 256;

	hipEventRecord(gpu_start, 0);

	find_minimum_kernel << < gridSize, blockSize >> > (d_array, d_min, d_mutex, N);
	hipMemcpy(h_min, d_min, sizeof(float) * 1, hipMemcpyDeviceToHost);
	
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

	std::cout << "Min number found on gpu was: " << *h_min << std::endl;
	std::cout << "The gpu took: " << gpu_elapsed_time / 1000 << " seconds" << std::endl;

	auto start = std::chrono::system_clock::now();
	*h_min = RAND_MAX;
	for (unsigned int i = 0; i < N; i++) {
		if (h_array[i] < *h_min) {
			*h_min = h_array[i];
		}
	}
	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	std::cout << "Min number found on cpu was: " << *h_min << std::endl;
	std::cout << "elapsed time: " << std::fixed << std::setprecision(20) << elapsed_seconds.count() << "s\n\n";

	free(h_array);
	free(h_min);
	hipFree(d_array);
	hipFree(d_min);
	hipFree(d_mutex);


	//	task 1
	N = 21 * pow(10,1);
	float step = float(21) / float(N);
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);

	find_foo_kernel << < gridSize, blockSize >> > (step, N);

	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);
	std::cout << "The gpu took: " << gpu_elapsed_time / 1000 << " seconds" << std::endl;

	start = std::chrono::system_clock::now();
	for (unsigned int i = 0; i < N; i++) {
		float x = 1 + float(i * step);
		float foo = 2 * (1 / tan(3 * x)) - (1 / (pow(12 * x, 2) + 7 * x - 5));
	}
	end = std::chrono::system_clock::now();
	elapsed_seconds = end - start;
	std::cout << "elapsed time: " << std::fixed << std::setprecision(20) << elapsed_seconds.count() << "s\n\n";


	system("pause");
}



